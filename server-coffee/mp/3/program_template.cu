#include "hip/hip_runtime.h"

#include	<wb.h>


__global__ void vecAdd(int * in, int * out, int width) {
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if (index < width)
        out[index] += in[index];
}

int main(void) {
	int ii;
	int * in, * out;
    int * d_in, * d_out;
    int width = 1<<10;

	wbTime_start(Generic, "Creating memory on host");
    in = (int *) malloc(width * sizeof(int));
    out = (int *) malloc(width * sizeof(int));
	wbTime_stop(Generic, "Creating memory on host");

    wbLog(TRACE, "HELLO Logger");

	wbTime_start(IO, "Initializing host values");
    for (ii = 0; ii < width; ii++) {
    	in[ii] = ii;
    	out[ii] = ii;
    }
	wbTime_stop(IO, "Initializing host values");

	wbTime_start(GPU, "Doing GPU allocation + computation");
    hipMalloc((void **) &d_in, width*sizeof(int));
    hipMalloc((void **) &d_out, width*sizeof(int));
    
    wbTime_start(Copy, "Copying memory to the device");
    hipMemcpy(d_in, in, width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, width * sizeof(int), hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying memory to the device");
    
    dim3 blockDim(32);
    dim3 gridDim(width/32);
    
    wbTime_start(Compute, "Performing CUDA computation");
    vecAdd<<<blockDim, gridDim>>>(d_in, d_out, width);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying memory back from the device");
    hipMemcpy(out, d_out, width * sizeof(int), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying memory back from the device");
	wbTime_stop(GPU, "Doing GPU allocation + computation");
    

    return 0;
}

